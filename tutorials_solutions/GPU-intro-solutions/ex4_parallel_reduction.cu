
#include <hip/hip_runtime.h>
#include <stdio.h>

void cpu_sum(int *x, int n)
{
    int result = 0;
    for(unsigned int i=0; i < n; ++i) { 
        result += x[i];
    }
    printf("CPU Sum is %d \n", result);
}

__global__ void sum(int *x)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  
    // printf("Hello from device! My threadId = blockIdx.x *blockDim.x + threadIdx.x <=> %d = %d * %d + %d \n",
    // tidx blockIdx.x, blockDim.x, threadIdx.x);

    // Reduction #1: Interleaved Addressing
    // for(unsigned int s=1; s < blockDim.x; s *= 2) {
    //     if (tid % (2*s) == 0) {
    //         printf("x[%d] = %d \n", tid,x[tid] );
    //         x[tid] += x[tid + s];
    //     }
    //     __syncthreads();
    // }
    
    // Reduction #2: Interleaved Addressing
    // int step  = 1;
    // while (step < blockDim.x)
    // {
    //     if (tid % step == 0)
    //     {   
    //         //printf("tid = %d \n", tid);
    //         x[tid] = x[tid] + x[tid+step];
    //     }   
    //     //__syncthreads();
    //     step *= 2;
    // }

    // Reduction #3: Sequential Addressing
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
        x[tid] += x[tid + s];
        }
        __syncthreads();
    }
}

int main()
{
    int h[] = {10, 1, 8, -1, 0, -2, 3, 5, -2, -3, 2, 7, 0, 11, 0, 2};
    
    int size = sizeof(h);
    int count = size/sizeof(int);
    
	int* d;
	hipMalloc((void**)&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    sum <<<1, count >>>(d);  // <<<blocks, threads_per_block>>>

    int result;
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);
    printf("GPU Sum is %d \n",result);
    
    cpu_sum(h, count);

    // int* processed_d   = (int*)malloc(size);
    // cudaMemcpy(processed_d, d, size, cudaMemcpyDeviceToHost);
    
    // for (int i = 0; i < count; ++i)
    //     printf("processed_d[%d]=%d \n", i, processed_d[i]);

    hipFree(d);
	return 0;
}
