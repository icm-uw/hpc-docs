#include <stdio.h>
#include <numeric>

#include "hip/hip_runtime.h"


using namespace std;

void cpu_sum(int *x, int n)
{
    int result = 0;
    for(unsigned int i=0; i < n; ++i) { 
        result += x[i];
    }
    printf("CPU result is %d \n", result);
}

__global__ void sum(int *x)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  
    // printf("Hello from device! My threadId = blockIdx.x *blockDim.x + threadIdx.x <=> %d = %d * %d + %d \n",
    // tidx blockIdx.x, blockDim.x, threadIdx.x);

    // for(unsigned int s=1; s < blockDim.x; s *= 2) {
    //     if (tid % (2*s) == 0) {
    //         printf("x[%d] = %d \n", tid,x[tid] );
    //         x[tid] += x[tid + s];
    //     }
    //     __syncthreads();
    // }
    
    int step  = 1;
    while (step < blockDim.x)
    {
        if (tid % step == 0)
        {   
            //printf("tid = %d \n", tid);
            x[tid] = x[tid] + x[tid+step];
        }   
        //__syncthreads();
        step *= 2;
    }
}

int main()
{
    int h[] = {10, 1, 8, -1, 0, -2, 3, 5, -2, -3, 2, 7, 0, 11, 0, 2};
    
    int size = sizeof(h);
    int count = size/sizeof(int);
    
	int* d;
	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    sum <<<1, count >>>(d);  // <<<blocks, threads_per_block>>>

    int result;
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);
    printf("GPU Sum is %d \n",result);
    
    cpu_sum(h, count);

    // int* processed_d   = (int*)malloc(size);
    // cudaMemcpy(processed_d, d, size, cudaMemcpyDeviceToHost);
    
    // for (int i = 0; i < count; ++i)
    //      cout << "processed_d[" << i << "] " << processed_d[i] << endl;

    hipFree(d);
	return 0;
}
