#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


//cudaMemcpy (void ∗dst, const void ∗src, size t count, enum cudaMemcpyKind kind)
#define MAX_THREADS_IN_BLOCK 1024

#define MAX_ERR 1e-6

using namespace std;

void cpu_vector_add(float *h_out, float *h_a, float *h_b, int n) {
    for(int i = 0; i < n; i++){
        h_out[i] = h_a[i] + h_b[i];
    }
}

__global__ void gpu_vector_add(float *out, float *a, float *b, int n) {
    // built-in variable blockDim.x describes amount threads per block

    // int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // if (tid < n)
    //     out[tid] = a[tid] + b[tid];
    
   
    // more advanced version - handling arbitrary vector/kernel size
    int i = blockIdx.x * blockDim.x + threadIdx.x;  
    int step = gridDim.x * blockDim.x;

    for(; i < n; i += step){
        out[i] = a[i] + b[i];
    }
}

void CPU_version_wrapper(const int N)
{
    float *h_a, *h_b, *h_out; 

    // Allocate memory
    h_a   = (float*)malloc(sizeof(float) * N);
    h_b   = (float*)malloc(sizeof(float) * N);
    h_out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        h_a[i] = 1.0; 
        h_b[i] = 2.0;
    }

    // Main function
    cpu_vector_add(h_out, h_a, h_b, N);

    for(int i = 0; i < N; i++){
        assert(fabs(h_out[i] - h_a[i] - h_b[i]) < MAX_ERR);
    }
    printf("CPU assertion PASSED\n");
    printf("CPU Last element in the array: out[%d] = %f\n\n",N-1,  h_out[N-1]);

    free(h_a);
    free(h_b);
    free(h_out);
}

void GPU_version_wrapper(const int N)
{

    // Allocate CPU memory
    float *h_a, *h_b, *h_out; 
    h_a   = (float*)malloc(sizeof(float) * N);
    h_b   = (float*)malloc(sizeof(float) * N);
    h_out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        h_a[i] = 1.0; 
        h_b[i] = 2.0;
    }

    // Allocate device memory for d_a
    float *d_a, *d_b, *d_out;
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, h_a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Main function
    // gpu_vector_add<<<1,1024>>>(d_out, d_a, d_b, N);

    gpu_vector_add<<<N/MAX_THREADS_IN_BLOCK,MAX_THREADS_IN_BLOCK>>>(d_out, d_a, d_b, N);

    // Transfer data from device memory to host
    hipMemcpy(h_out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    // cudaMemcpy() Blocks the CPU until the copy is complete
    // Copy begins when all preceding CUDA calls have completed

    // Verification
    printf("GPU Last element in the array: out[%d] = %f\n",N-1,  h_out[N-1]);
    for(int i = 0; i < N; i++){
        assert(fabs(h_out[i] - h_a[i] - h_b[i]) < MAX_ERR);
    }

    printf("GPU assertion PASSED\n\n");
    
    // Cleanup memory after kernel execution
    hipFree(d_a);hipFree(d_b);hipFree(d_out);
    free(h_a);free(h_b);free(h_out);
}



int main(){
    const int N = 10240;
    CPU_version_wrapper(N);
    GPU_version_wrapper(N);

    return 0;
}


// experiment with inappriopate d_array by threads adressing

// cuda-memcheck ./ex2_vector_add 
// ========= CUDA-MEMCHECK
// CPU Result: 3
// ========= Invalid __global__ read of size 4
// =========     at 0x00000050 in gpu_vector_add(float*, float*, float*, int)
// =========     by thread (10,0,0) in block (9,0,0)
// =========     Address 0xb021c01b4 is out of bounds

//https://devtalk.nvidia.com/default/topic/978550/cuda-programming-and-performance/maximum-number-of-threads-on-thread-block/