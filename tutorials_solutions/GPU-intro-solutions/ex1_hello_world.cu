

#include <hip/hip_runtime.h>
#include <iostream>

// functions qualifers:
// __global__ launched by CPU on device (must return void)
// __device__ called from other GPU functions (never CPU)
// __host__ can be executed by CPU
// (can be used together with __device__)

// kernel launch:
// f_name<<<gridDim, blockDim, sharedMem, strId>>>(p1,... pN)

__global__ void print_from_gpu(void) {
    printf("Hello from device! I am threadIdx.x %d in blockIdx.x %d, the blockDim.x is %d \n",
    threadIdx.x, blockIdx.x, blockDim.x);
    printf("Hello from device! I am threadIdx.y %d in blockIdx.y %d, the blockDim.y is %d \n",
    threadIdx.y, blockIdx.y, blockDim.y);
    printf("Hello from device! I am threadIdx.z %d in blockIdx.z %d, the blockDim.z is %d \n",
    threadIdx.z, blockIdx.z, blockDim.z);
}

int main(void) {
    printf("Hello World from host!\n");

    print_from_gpu<<<1,3>>>();  // <<<blocks, threads_per_block>>>

    hipDeviceSynchronize();
    return 0;
}